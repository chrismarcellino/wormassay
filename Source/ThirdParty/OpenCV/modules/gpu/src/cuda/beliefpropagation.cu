#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "opencv2/gpu/devmem2d.hpp"
#include "saturate_cast.hpp"
#include "safe_call.hpp"

using namespace cv::gpu;

#ifndef FLT_MAX
#define FLT_MAX 3.402823466e+38F
#endif

namespace cv { namespace gpu { namespace bp {

///////////////////////////////////////////////////////////////
/////////////////////// load constants ////////////////////////
///////////////////////////////////////////////////////////////

    __constant__ int   cndisp;
    __constant__ float cmax_data_term;
    __constant__ float cdata_weight;
    __constant__ float cmax_disc_term;
    __constant__ float cdisc_single_jump;

    void load_constants(int ndisp, float max_data_term, float data_weight, float max_disc_term, float disc_single_jump)
    {
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cndisp),            &ndisp,            sizeof(int  )) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_data_term),    &max_data_term,    sizeof(float)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdata_weight),      &data_weight,      sizeof(float)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cmax_disc_term),    &max_disc_term,    sizeof(float)) );
        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cdisc_single_jump), &disc_single_jump, sizeof(float)) );         
    }

///////////////////////////////////////////////////////////////
////////////////////////// comp data //////////////////////////
///////////////////////////////////////////////////////////////


    template <typename T>
    __global__ void comp_data_gray(const uchar* l, const uchar* r, size_t step, T* data, size_t data_step, int cols, int rows) 
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y > 0 && y < rows - 1 && x > 0 && x < cols - 1)
        {
            const uchar* ls = l + y * step + x; 
            const uchar* rs = r + y * step + x; 

            T* ds = data + y * data_step + x;
            size_t disp_step = data_step * rows;

            for (int disp = 0; disp < cndisp; disp++) 
            {
                if (x - disp >= 1)
                {
                    float val  = abs((int)ls[0] - rs[-disp]);
                    
                    ds[disp * disp_step] = saturate_cast<T>(fmin(cdata_weight * val, cdata_weight * cmax_data_term));
                }
                else
                {
                    ds[disp * disp_step] = saturate_cast<T>(cdata_weight * cmax_data_term);
                }
            }
        }
    }

    template <typename T>
    __global__ void comp_data_bgr(const uchar* l, const uchar* r, size_t step, T* data, size_t data_step, int cols, int rows) 
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y > 0 && y < rows - 1 && x > 0 && x < cols - 1)
        {
            const uchar* ls = l + y * step + x * 3; 
            const uchar* rs = r + y * step + x * 3; 

            T* ds = data + y * data_step + x;
            size_t disp_step = data_step * rows;

            for (int disp = 0; disp < cndisp; disp++) 
            {
                if (x - disp >= 1)
                {                    
                    const float tr = 0.299f;
                    const float tg = 0.587f;
                    const float tb = 0.114f;

                    float val  = tb * abs((int)ls[0] - rs[0-disp*3]);
                          val += tg * abs((int)ls[1] - rs[1-disp*3]);
                          val += tr * abs((int)ls[2] - rs[2-disp*3]);
                    
                    ds[disp * disp_step] = saturate_cast<T>(fmin(cdata_weight * val, cdata_weight * cmax_data_term));
                }
                else
                {
                    ds[disp * disp_step] = saturate_cast<T>(cdata_weight * cmax_data_term);
                }
            }
        }
    }

    typedef void (*CompDataFunc)(const DevMem2D& l, const DevMem2D& r, int channels, DevMem2D mdata, const hipStream_t& stream);

    template<typename T>
    void comp_data_(const DevMem2D& l, const DevMem2D& r, int channels, DevMem2D mdata, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(l.cols, threads.x);
        grid.y = divUp(l.rows, threads.y);
        
        if (channels == 1)
            comp_data_gray<T><<<grid, threads, 0, stream>>>(l.data, r.data, l.step, (T*)mdata.data, mdata.step/sizeof(T), l.cols, l.rows);
        else
            comp_data_bgr<T><<<grid, threads, 0, stream>>>(l.data, r.data, l.step, (T*)mdata.data, mdata.step/sizeof(T), l.cols, l.rows);
        
        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void comp_data(int msg_type, const DevMem2D& l, const DevMem2D& r, int channels, DevMem2D mdata, const hipStream_t& stream)
    {
        static CompDataFunc tab[8] =
        {
            0,                  // uchar
            0,                  // schar
            0,                  // ushort
            comp_data_<short>,  // short
            0,                  // int
            comp_data_<float>,  // float
            0,                  // double
            0                   // user type
        };

        CompDataFunc func = tab[msg_type];
        if (func == 0)
            cv::gpu::error("Unsupported message type", __FILE__, __LINE__);
        func(l, r, channels, mdata, stream);
    }

///////////////////////////////////////////////////////////////
//////////////////////// data step down ///////////////////////
///////////////////////////////////////////////////////////////

    template <typename T>
    __global__ void data_step_down(int dst_cols, int dst_rows, int src_rows, const T* src, size_t src_step, T* dst, size_t dst_step)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < dst_cols && y < dst_rows)
        {
            const size_t dst_disp_step = dst_step * dst_rows;
            const size_t src_disp_step = src_step * src_rows;

            for (int d = 0; d < cndisp; ++d)
            {
                float dst_reg  = src[d * src_disp_step + src_step * (2*y+0) + (2*x+0)];
                      dst_reg += src[d * src_disp_step + src_step * (2*y+1) + (2*x+0)];
                      dst_reg += src[d * src_disp_step + src_step * (2*y+0) + (2*x+1)];
                      dst_reg += src[d * src_disp_step + src_step * (2*y+1) + (2*x+1)];

                dst[d * dst_disp_step + y * dst_step + x] = saturate_cast<T>(dst_reg);
            }
        }
    }

    typedef void (*DataStepDownFunc)(int dst_cols, int dst_rows, int src_rows, const DevMem2D& src, DevMem2D dst, const hipStream_t& stream);

    template<typename T>
    void data_step_down_(int dst_cols, int dst_rows, int src_rows, const DevMem2D& src, DevMem2D dst, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(dst_cols, threads.x);
        grid.y = divUp(dst_rows, threads.y);

        data_step_down<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (const T*)src.data, src.step/sizeof(T), (T*)dst.data, dst.step/sizeof(T));
        
        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void data_step_down(int dst_cols, int dst_rows, int src_rows, int msg_type, const DevMem2D& src, DevMem2D dst, const hipStream_t& stream)
    {
        static DataStepDownFunc tab[8] =
        {
            0,                       // uchar
            0,                       // schar
            0,                       // ushort
            data_step_down_<short>,  // short
            0,                       // int
            data_step_down_<float>,  // float
            0,                       // double
            0                        // user type
        };

        DataStepDownFunc func = tab[msg_type];
        if (func == 0)
            cv::gpu::error("Unsupported message type", __FILE__, __LINE__);
        func(dst_cols, dst_rows, src_rows, src, dst, stream);
    }

///////////////////////////////////////////////////////////////
/////////////////// level up messages  ////////////////////////
///////////////////////////////////////////////////////////////

    template <typename T>
    __global__ void level_up_message(int dst_cols, int dst_rows, int src_rows, const T* src, size_t src_step, T* dst, size_t dst_step)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;        

        if (x < dst_cols && y < dst_rows)
        {
            const size_t dst_disp_step = dst_step * dst_rows;
            const size_t src_disp_step = src_step * src_rows;

            T*       dstr = dst + y   * dst_step + x;
            const T* srcr = src + y/2 * src_step + x/2;

            for (int d = 0; d < cndisp; ++d)            
                dstr[d * dst_disp_step] = srcr[d * src_disp_step];
        }
    }

    typedef void (*LevelUpMessagesFunc)(int dst_idx, int dst_cols, int dst_rows, int src_rows, DevMem2D* mus, DevMem2D* mds, DevMem2D* mls, DevMem2D* mrs, const hipStream_t& stream);

    template<typename T>
    void level_up_messages_(int dst_idx, int dst_cols, int dst_rows, int src_rows, DevMem2D* mus, DevMem2D* mds, DevMem2D* mls, DevMem2D* mrs, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(dst_cols, threads.x);
        grid.y = divUp(dst_rows, threads.y);

        int src_idx = (dst_idx + 1) & 1;

        level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (const T*)mus[src_idx].data, mus[src_idx].step/sizeof(T), (T*)mus[dst_idx].data, mus[dst_idx].step/sizeof(T));
        level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (const T*)mds[src_idx].data, mds[src_idx].step/sizeof(T), (T*)mds[dst_idx].data, mds[dst_idx].step/sizeof(T));
        level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (const T*)mls[src_idx].data, mls[src_idx].step/sizeof(T), (T*)mls[dst_idx].data, mls[dst_idx].step/sizeof(T));
        level_up_message<T><<<grid, threads, 0, stream>>>(dst_cols, dst_rows, src_rows, (const T*)mrs[src_idx].data, mrs[src_idx].step/sizeof(T), (T*)mrs[dst_idx].data, mrs[dst_idx].step/sizeof(T));
        
        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void level_up_messages(int dst_idx, int dst_cols, int dst_rows, int src_rows, int msg_type, DevMem2D* mus, DevMem2D* mds, DevMem2D* mls, DevMem2D* mrs, const hipStream_t& stream)
    {
        static LevelUpMessagesFunc tab[8] =
        {
            0,                          // uchar
            0,                          // schar
            0,                          // ushort
            level_up_messages_<short>,  // short
            0,                          // int
            level_up_messages_<float>,  // float
            0,                          // double
            0                           // user type
        };

        LevelUpMessagesFunc func = tab[msg_type];
        if (func == 0)
            cv::gpu::error("Unsupported message type", __FILE__, __LINE__);
        func(dst_idx, dst_cols, dst_rows, src_rows, mus, mds, mls, mrs, stream);
    }

///////////////////////////////////////////////////////////////
////////////////////  calc all iterations /////////////////////
///////////////////////////////////////////////////////////////

    template <typename T>
    __device__ void calc_min_linear_penalty(T* dst, size_t step)
    {
        float prev = dst[0];
        float cur;
        for (int disp = 1; disp < cndisp; ++disp) 
        {
            prev += cdisc_single_jump;
            cur = dst[step * disp];
            if (prev < cur)
            {
                cur = prev;
                dst[step * disp] = saturate_cast<T>(prev);
            }
            prev = cur;
        }

        prev = dst[(cndisp - 1) * step];
        for (int disp = cndisp - 2; disp >= 0; disp--)     
        {
            prev += cdisc_single_jump;
            cur = dst[step * disp];
            if (prev < cur)
            {
                cur = prev;
                dst[step * disp] = saturate_cast<T>(prev);
            }
            prev = cur;      
        }
    }

    template <typename T>
    __device__ void message(const T* msg1, const T* msg2, const T* msg3, const T* data, T* dst, size_t msg_disp_step, size_t data_disp_step)
    {
        float minimum = FLT_MAX;

        for(int i = 0; i < cndisp; ++i)
        {
            float dst_reg  = msg1[msg_disp_step * i];
                  dst_reg += msg2[msg_disp_step * i];
                  dst_reg += msg3[msg_disp_step * i];
                  dst_reg += data[data_disp_step * i];

            if (dst_reg < minimum)
                minimum = dst_reg;

            dst[msg_disp_step * i] = saturate_cast<T>(dst_reg);
        }

        calc_min_linear_penalty(dst, msg_disp_step);

        minimum += cmax_disc_term;

        float sum = 0;
        for(int i = 0; i < cndisp; ++i)
        {
            float dst_reg = dst[msg_disp_step * i];
            if (dst_reg > minimum)
            {
                dst_reg = minimum;
                dst[msg_disp_step * i] = saturate_cast<T>(minimum);
            }
            sum += dst_reg;
        }    
        sum /= cndisp;

        for(int i = 0; i < cndisp; ++i)
            dst[msg_disp_step * i] -= sum;
    }

    template <typename T>
    __global__ void one_iteration(int t, T* u, T* d, T* l, T* r, size_t msg_step, const T* data, size_t data_step, int cols, int rows)
    {
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int x = ((blockIdx.x * blockDim.x + threadIdx.x) << 1) + ((y + t) & 1);

        if ( (y > 0) && (y < rows - 1) && (x > 0) && (x < cols - 1))
        {
            T* us = u + y * msg_step + x;
            T* ds = d + y * msg_step + x;
            T* ls = l + y * msg_step + x;
            T* rs = r + y * msg_step + x;
            const T* dt = data + y * data_step + x;

            size_t msg_disp_step = msg_step * rows;
            size_t data_disp_step = data_step * rows;

            message(us + msg_step, ls        + 1, rs - 1, dt, us, msg_disp_step, data_disp_step);
            message(ds - msg_step, ls        + 1, rs - 1, dt, ds, msg_disp_step, data_disp_step);
            message(us + msg_step, ds - msg_step, rs - 1, dt, rs, msg_disp_step, data_disp_step);
            message(us + msg_step, ds - msg_step, ls + 1, dt, ls, msg_disp_step, data_disp_step);                
        }
    }

    typedef void (*CalcAllIterationFunc)(int cols, int rows, int iters, DevMem2D& u, DevMem2D& d, DevMem2D& l, DevMem2D& r, const DevMem2D& data, const hipStream_t& stream);

    template<typename T>
    void calc_all_iterations_(int cols, int rows, int iters, DevMem2D& u, DevMem2D& d, DevMem2D& l, DevMem2D& r, const DevMem2D& data, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(cols, threads.x << 1);
        grid.y = divUp(rows, threads.y);

        for(int t = 0; t < iters; ++t)
        {
            one_iteration<T><<<grid, threads, 0, stream>>>(t, (T*)u.data, (T*)d.data, (T*)l.data, (T*)r.data, u.step/sizeof(T), (const T*)data.data, data.step/sizeof(T), cols, rows);
            
            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }
    }

    void calc_all_iterations(int cols, int rows, int iters, int msg_type, DevMem2D& u, DevMem2D& d, DevMem2D& l, DevMem2D& r, const DevMem2D& data, const hipStream_t& stream)
    {
        static CalcAllIterationFunc tab[8] =
        {
            0,                            // uchar
            0,                            // schar
            0,                            // ushort
            calc_all_iterations_<short>,  // short
            0,                            // int
            calc_all_iterations_<float>,  // float
            0,                            // double
            0                             // user type
        };

        CalcAllIterationFunc func = tab[msg_type];
        if (func == 0)
            cv::gpu::error("Unsupported message type", __FILE__, __LINE__);
        func(cols, rows, iters, u, d, l, r, data, stream);
    }

///////////////////////////////////////////////////////////////
/////////////////////////// output ////////////////////////////
///////////////////////////////////////////////////////////////

    template <typename T>
    __global__ void output(int cols, int rows, const T* u, const T* d, const T* l, const T* r, const T* data, size_t step, short* disp, size_t res_step) 
    {   
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y > 0 && y < rows - 1 && x > 0 && x < cols - 1)
        {
            const T* us = u + (y + 1) * step + x;
            const T* ds = d + (y - 1) * step + x;
            const T* ls = l + y * step + (x + 1);
            const T* rs = r + y * step + (x - 1);
            const T* dt = data + y * step + x;

            size_t disp_step = rows * step;

            int best = 0;
            float best_val = FLT_MAX;
            for (int d = 0; d < cndisp; ++d) 
            {
                float val  = us[d * disp_step];
                      val += ds[d * disp_step];
                      val += ls[d * disp_step];
                      val += rs[d * disp_step];
                      val += dt[d * disp_step];

                if (val < best_val) 
                {
                    best_val = val;
                    best = d;
                }
            }

            disp[res_step * y + x] = saturate_cast<short>(best);
        }
    }

    typedef void (*OutputFunc)(const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data, DevMem2D disp, const hipStream_t& stream);

    template<typename T>
    void output_(const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data, DevMem2D disp, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(disp.cols, threads.x);
        grid.y = divUp(disp.rows, threads.y);

        output<T><<<grid, threads, 0, stream>>>(disp.cols, disp.rows, (const T*)u.data, (const T*)d.data, (const T*)l.data, (const T*)r.data, (const T*)data.data, u.step/sizeof(T), (short*)disp.data, disp.step/sizeof(short));

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void output(int msg_type, const DevMem2D& u, const DevMem2D& d, const DevMem2D& l, const DevMem2D& r, const DevMem2D& data, DevMem2D disp, const hipStream_t& stream)
    {            
        static OutputFunc tab[8] =
        {
            0,               // uchar
            0,               // schar
            0,               // ushort
            output_<short>,  // short
            0,               // int
            output_<float>,  // float
            0,               // double
            0                // user type
        };

        OutputFunc func = tab[msg_type];
        if (func == 0)
            cv::gpu::error("Unsupported message type", __FILE__, __LINE__);
        func(u, d, l, r, data, disp, stream);
    }

}}}