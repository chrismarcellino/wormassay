#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"
#include "transform.hpp"
#include "limits_gpu.hpp"

using namespace cv::gpu;
using namespace cv::gpu::device;

#ifndef CV_PI
#define CV_PI   3.1415926535897932384626433832795f
#endif

//////////////////////////////////////////////////////////////////////////////////////
// Cart <-> Polar

namespace cv { namespace gpu { namespace mathfunc
{
    struct Nothing
    {
        static __device__ void calc(int, int, float, float, float*, size_t, float)
        {
        }
    };
    struct Magnitude
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
        {
            dst[y * dst_step + x] = sqrtf(x_data * x_data + y_data * y_data);
        }
    };
    struct MagnitudeSqr
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float)
        {
            dst[y * dst_step + x] = x_data * x_data + y_data * y_data;
        }
    };
    struct Atan2
    {
        static __device__ void calc(int x, int y, float x_data, float y_data, float* dst, size_t dst_step, float scale)
        {
            dst[y * dst_step + x] = scale * atan2f(y_data, x_data);
        }
    };
    template <typename Mag, typename Angle>
    __global__ void cartToPolar(const float* xptr, size_t x_step, const float* yptr, size_t y_step, 
                                float* mag, size_t mag_step, float* angle, size_t angle_step, float scale, int width, int height)
    {
		const int x = blockDim.x * blockIdx.x + threadIdx.x;
		const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            float x_data = xptr[y * x_step + x];
            float y_data = yptr[y * y_step + x];

            Mag::calc(x, y, x_data, y_data, mag, mag_step, scale);
            Angle::calc(x, y, x_data, y_data, angle, angle_step, scale);
        }
    }

    struct NonEmptyMag
    {
        static __device__ float get(const float* mag, size_t mag_step, int x, int y)
        {
            return mag[y * mag_step + x];
        }
    };
    struct EmptyMag
    {
        static __device__ float get(const float*, size_t, int, int)
        {
            return 1.0f;
        }
    };
    template <typename Mag>
    __global__ void polarToCart(const float* mag, size_t mag_step, const float* angle, size_t angle_step, float scale,
        float* xptr, size_t x_step, float* yptr, size_t y_step, int width, int height)
    {
		const int x = blockDim.x * blockIdx.x + threadIdx.x;
		const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            float mag_data = Mag::get(mag, mag_step, x, y);
            float angle_data = angle[y * angle_step + x];
            float sin_a, cos_a;

            sincosf(scale * angle_data, &sin_a, &cos_a);

            xptr[y * x_step + x] = mag_data * cos_a;
            yptr[y * y_step + x] = mag_data * sin_a;
        }
    }

    template <typename Mag, typename Angle>
    void cartToPolar_caller(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(x.cols, threads.x);
        grid.y = divUp(x.rows, threads.y);
        
        const float scale = angleInDegrees ? (float)(180.0f / CV_PI) : 1.f;

        cartToPolar<Mag, Angle><<<grid, threads, 0, stream>>>(
            x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), 
            mag.data, mag.step/mag.elemSize(), angle.data, angle.step/angle.elemSize(), scale, x.cols, x.rows);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void cartToPolar_gpu(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, bool magSqr, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2Df& x, const DevMem2Df& y, const DevMem2Df& mag, const DevMem2Df& angle, bool angleInDegrees, hipStream_t stream);
        static const caller_t callers[2][2][2] = 
        {
            {
                {
                    cartToPolar_caller<Magnitude, Atan2>,
                    cartToPolar_caller<Magnitude, Nothing>
                },
                {
                    cartToPolar_caller<MagnitudeSqr, Atan2>,
                    cartToPolar_caller<MagnitudeSqr, Nothing>,
                }
            },
            {
                {
                    cartToPolar_caller<Nothing, Atan2>,
                    cartToPolar_caller<Nothing, Nothing>
                },
                {
                    cartToPolar_caller<Nothing, Atan2>,
                    cartToPolar_caller<Nothing, Nothing>,
                }
            }
        };

        callers[mag.data == 0][magSqr][angle.data == 0](x, y, mag, angle, angleInDegrees, stream);
    }

    template <typename Mag>
    void polarToCart_caller(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);

        grid.x = divUp(mag.cols, threads.x);
        grid.y = divUp(mag.rows, threads.y);
        
        const float scale = angleInDegrees ? (float)(CV_PI / 180.0f) : 1.0f;

        polarToCart<Mag><<<grid, threads, 0, stream>>>(mag.data, mag.step/mag.elemSize(), 
            angle.data, angle.step/angle.elemSize(), scale, x.data, x.step/x.elemSize(), y.data, y.step/y.elemSize(), mag.cols, mag.rows);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void polarToCart_gpu(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream)
    {
        typedef void (*caller_t)(const DevMem2Df& mag, const DevMem2Df& angle, const DevMem2Df& x, const DevMem2Df& y, bool angleInDegrees, hipStream_t stream);
        static const caller_t callers[2] = 
        {
            polarToCart_caller<NonEmptyMag>,
            polarToCart_caller<EmptyMag>
        };

        callers[mag.data == 0](mag, angle, x, y, angleInDegrees, stream);
    }

//////////////////////////////////////////////////////////////////////////////////////
// Compare

    template <typename T1, typename T2>
    struct NotEqual
    {
        __device__ uchar operator()(const T1& src1, const T2& src2)
        {
            return static_cast<uchar>(static_cast<int>(src1 != src2) * 255);
        }
    };

    template <typename T1, typename T2>
    inline void compare_ne(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        NotEqual<T1, T2> op;
        transform(static_cast< DevMem2D_<T1> >(src1), static_cast< DevMem2D_<T2> >(src2), dst, op, 0);
    }

    void compare_ne_8uc4(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<uint, uint>(src1, src2, dst);
    }
    void compare_ne_32f(const DevMem2D& src1, const DevMem2D& src2, const DevMem2D& dst)
    {
        compare_ne<float, float>(src1, src2, dst);
    }


//////////////////////////////////////////////////////////////////////////////
// Per-element bit-wise logical matrix operations

    struct Mask8U
    {
        explicit Mask8U(PtrStep mask): mask(mask) {}
        __device__ bool operator()(int y, int x) const { return mask.ptr(y)[x]; }
        PtrStep mask;
    };
    struct MaskTrue { __device__ bool operator()(int y, int x) const { return true; } };

    // Unary operations

    enum { UN_OP_NOT };

    template <typename T, int opid>
    struct UnOp { __device__ T operator()(T lhs, T rhs); };

    template <typename T>
    struct UnOp<T, UN_OP_NOT>{ __device__ T operator()(T x) { return ~x; } };

    template <typename T, int cn, typename UnOp, typename Mask>
    __global__ void bitwise_un_op(int rows, int cols, const PtrStep src, PtrStep dst, UnOp op, Mask mask)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask(y, x)) 
        {
            T* dsty = (T*)dst.ptr(y);
            const T* srcy = (const T*)src.ptr(y);

            #pragma unroll
            for (int i = 0; i < cn; ++i)
                dsty[cn * x + i] = op(srcy[cn * x + i]);
        }
    }

    template <int opid, typename Mask>
    void bitwise_un_op(int rows, int cols, const PtrStep src, PtrStep dst, int elem_size, Mask mask, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
        switch (elem_size)
        {
        case 1: bitwise_un_op<unsigned char, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned char, opid>(), mask); break;
        case 2: bitwise_un_op<unsigned short, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned short, opid>(), mask); break;
        case 3: bitwise_un_op<unsigned char, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned char, opid>(), mask); break;
        case 4: bitwise_un_op<unsigned int, 1><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 6: bitwise_un_op<unsigned short, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned short, opid>(), mask); break;
        case 8: bitwise_un_op<unsigned int, 2><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;       
        case 12: bitwise_un_op<unsigned int, 3><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 16: bitwise_un_op<unsigned int, 4><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 24: bitwise_un_op<unsigned int, 6><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        case 32: bitwise_un_op<unsigned int, 8><<<grid, threads>>>(rows, cols, src, dst, UnOp<unsigned int, opid>(), mask); break;
        }
        if (stream == 0) cudaSafeCall(hipDeviceSynchronize());        
    }

    void bitwise_not_caller(int rows, int cols,const PtrStep src, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_un_op<UN_OP_NOT>(rows, cols, src, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_not_caller(int rows, int cols,const PtrStep src, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_un_op<UN_OP_NOT>(rows, cols, src, dst, elem_size, Mask8U(mask), stream);
    }

    // Binary operations

    enum { BIN_OP_OR, BIN_OP_AND, BIN_OP_XOR };

    template <typename T, int opid>
    struct BinOp { __device__ T operator()(T lhs, T rhs); };

    template <typename T>
    struct BinOp<T, BIN_OP_OR>{ __device__ T operator()(T lhs, T rhs) { return lhs | rhs; } };

    template <typename T>
    struct BinOp<T, BIN_OP_AND>{ __device__ T operator()(T lhs, T rhs) { return lhs & rhs; } };

    template <typename T>
    struct BinOp<T, BIN_OP_XOR>{ __device__ T operator()(T lhs, T rhs) { return lhs ^ rhs; } };

    template <typename T, int cn, typename BinOp, typename Mask>
    __global__ void bitwise_bin_op(int rows, int cols, const PtrStep src1, const PtrStep src2, PtrStep dst, BinOp op, Mask mask)
    {
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < cols && y < rows && mask(y, x)) 
        {
            T* dsty = (T*)dst.ptr(y);
            const T* src1y = (const T*)src1.ptr(y);
            const T* src2y = (const T*)src2.ptr(y);

            #pragma unroll
            for (int i = 0; i < cn; ++i)
                dsty[cn * x + i] = op(src1y[cn * x + i], src2y[cn * x + i]);
        }
    }

    template <int opid, typename Mask>
    void bitwise_bin_op(int rows, int cols, const PtrStep src1, const PtrStep src2, PtrStep dst, int elem_size, Mask mask, hipStream_t stream)
    {
        dim3 threads(16, 16);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));
        switch (elem_size)
        {
        case 1: bitwise_bin_op<unsigned char, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned char, opid>(), mask); break;
        case 2: bitwise_bin_op<unsigned short, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned short, opid>(), mask); break;
        case 3: bitwise_bin_op<unsigned char, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned char, opid>(), mask); break;
        case 4: bitwise_bin_op<unsigned int, 1><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 6: bitwise_bin_op<unsigned short, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned short, opid>(), mask); break;
        case 8: bitwise_bin_op<unsigned int, 2><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;       
        case 12: bitwise_bin_op<unsigned int, 3><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 16: bitwise_bin_op<unsigned int, 4><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 24: bitwise_bin_op<unsigned int, 6><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        case 32: bitwise_bin_op<unsigned int, 8><<<grid, threads>>>(rows, cols, src1, src2, dst, BinOp<unsigned int, opid>(), mask); break;
        }
        if (stream == 0) cudaSafeCall(hipDeviceSynchronize());        
    }

    void bitwise_or_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_OR>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_or_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_OR>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }

    void bitwise_and_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_AND>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_and_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_AND>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }

    void bitwise_xor_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_XOR>(rows, cols, src1, src2, dst, elem_size, MaskTrue(), stream);
    }

    void bitwise_xor_caller(int rows, int cols, const PtrStep src1, const PtrStep src2, int elem_size, PtrStep dst, const PtrStep mask, hipStream_t stream)
    {
        bitwise_bin_op<BIN_OP_XOR>(rows, cols, src1, src2, dst, elem_size, Mask8U(mask), stream);
    }  



//////////////////////////////////////////////////////////////////////////////
// Min max

    // To avoid shared bank conflicts we convert each value into value of 
    // appropriate type (32 bits minimum)
    template <typename T> struct MinMaxTypeTraits {};
    template <> struct MinMaxTypeTraits<unsigned char> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<char> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<unsigned short> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<short> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<int> { typedef int best_type; };
    template <> struct MinMaxTypeTraits<float> { typedef float best_type; };
    template <> struct MinMaxTypeTraits<double> { typedef double best_type; };


    namespace minmax 
    {

    __constant__ int ctwidth;
    __constant__ int ctheight;

    // Global counter of blocks finished its work
    __device__ unsigned int blocks_finished = 0;


    // Estimates good thread configuration
    //  - threads variable satisfies to threads.x * threads.y == 256
    void estimate_thread_cfg(int cols, int rows, dim3& threads, dim3& grid)
    {
        threads = dim3(32, 8);
        grid = dim3(divUp(cols, threads.x * 8), divUp(rows, threads.y * 32));
    }


    // Returns required buffer sizes
    void get_buf_size_required(int cols, int rows, int elem_size, int& bufcols, int& bufrows)
    {
        dim3 threads, grid;
        estimate_thread_cfg(cols, rows, threads, grid);
        bufcols = grid.x * grid.y * elem_size; 
        bufrows = 2;
    }


    // Estimates device constants which are used in the kernels using specified thread configuration
    void set_kernel_consts(int cols, int rows, const dim3& threads, const dim3& grid)
    {        
        int twidth = divUp(divUp(cols, grid.x), threads.x);
        int theight = divUp(divUp(rows, grid.y), threads.y);
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctwidth), &twidth, sizeof(ctwidth))); 
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctheight), &theight, sizeof(ctheight))); 
    }  


    // Does min and max in shared memory
    template <typename T>
    __device__ void merge(unsigned int tid, unsigned int offset, volatile T* minval, volatile T* maxval)
    {
        minval[tid] = min(minval[tid], minval[tid + offset]);
        maxval[tid] = max(maxval[tid], maxval[tid + offset]);
    }


    template <int size, typename T>
    __device__ void find_min_max_in_smem(volatile T* minval, volatile T* maxval, const unsigned int tid)
    {
        if (size >= 512) { if (tid < 256) { merge(tid, 256, minval, maxval); } __syncthreads(); }
        if (size >= 256) { if (tid < 128) { merge(tid, 128, minval, maxval); }  __syncthreads(); }
        if (size >= 128) { if (tid < 64) { merge(tid, 64, minval, maxval); } __syncthreads(); }

        if (tid < 32)
        {
            if (size >= 64) merge(tid, 32, minval, maxval);
            if (size >= 32) merge(tid, 16, minval, maxval);
            if (size >= 16) merge(tid, 8, minval, maxval);
            if (size >= 8) merge(tid, 4, minval, maxval);
            if (size >= 4) merge(tid, 2, minval, maxval);
            if (size >= 2) merge(tid, 1, minval, maxval);
        }
    }


    template <int nthreads, typename T, typename Mask>
    __global__ void min_max_kernel(const DevMem2D src, Mask mask, T* minval, T* maxval)
    {
        typedef typename MinMaxTypeTraits<T>::best_type best_type;
        __shared__ best_type sminval[nthreads];
        __shared__ best_type smaxval[nthreads];

        unsigned int x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
        unsigned int y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        T mymin = numeric_limits_gpu<T>::max();
        T mymax = numeric_limits_gpu<T>::is_signed ? -numeric_limits_gpu<T>::max() : numeric_limits_gpu<T>::min();
        unsigned int y_end = min(y0 + (ctheight - 1) * blockDim.y + 1, src.rows);
        unsigned int x_end = min(x0 + (ctwidth - 1) * blockDim.x + 1, src.cols);
        for (unsigned int y = y0; y < y_end; y += blockDim.y)
        {
            const T* src_row = (const T*)src.ptr(y);
            for (unsigned int x = x0; x < x_end; x += blockDim.x)
            {
                T val = src_row[x];
                if (mask(y, x)) 
                { 
                    mymin = min(mymin, val); 
                    mymax = max(mymax, val); 
                }
            }
        }

        sminval[tid] = mymin;
        smaxval[tid] = mymax;
        __syncthreads();

        find_min_max_in_smem<nthreads, best_type>(sminval, smaxval, tid);

        if (tid == 0) 
        {
            minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
            maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
        }

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 110
		__shared__ bool is_last;

		if (tid == 0)
		{
			minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
            maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
			__threadfence();

			unsigned int ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
			is_last = ticket == gridDim.x * gridDim.y - 1;
		}

		__syncthreads();

		if (is_last)
		{
            unsigned int idx = min(tid, gridDim.x * gridDim.y - 1);

            sminval[tid] = minval[idx];
            smaxval[tid] = maxval[idx];
            __syncthreads();

			find_min_max_in_smem<nthreads, best_type>(sminval, smaxval, tid);

            if (tid == 0) 
            {
                minval[0] = (T)sminval[0];
                maxval[0] = (T)smaxval[0];
                blocks_finished = 0;
            }
		}
#else
        if (tid == 0) 
        {
            minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
            maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
        }
#endif
    }

   
    template <typename T>
    void min_max_mask_caller(const DevMem2D src, const PtrStep mask, double* minval, double* maxval, PtrStep buf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        T* minval_buf = (T*)buf.ptr(0);
        T* maxval_buf = (T*)buf.ptr(1);

        min_max_kernel<256, T, Mask8U><<<grid, threads>>>(src, Mask8U(mask), minval_buf, maxval_buf);
        cudaSafeCall(hipDeviceSynchronize());

        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;
    }  

    template void min_max_mask_caller<unsigned char>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_caller<char>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_caller<unsigned short>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_caller<short>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_caller<int>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_caller<float>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_caller<double>(const DevMem2D, const PtrStep, double*, double*, PtrStep);


    template <typename T>
    void min_max_caller(const DevMem2D src, double* minval, double* maxval, PtrStep buf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        T* minval_buf = (T*)buf.ptr(0);
        T* maxval_buf = (T*)buf.ptr(1);

        min_max_kernel<256, T, MaskTrue><<<grid, threads>>>(src, MaskTrue(), minval_buf, maxval_buf);
        cudaSafeCall(hipDeviceSynchronize());

        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;
    }  

    template void min_max_caller<unsigned char>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_caller<char>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_caller<unsigned short>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_caller<short>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_caller<int>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_caller<float>(const DevMem2D, double*,double*, PtrStep);
    template void min_max_caller<double>(const DevMem2D, double*, double*, PtrStep);


    template <int nthreads, typename T>
    __global__ void min_max_pass2_kernel(T* minval, T* maxval, int size)
    {
        typedef typename MinMaxTypeTraits<T>::best_type best_type;
        __shared__ best_type sminval[nthreads];
        __shared__ best_type smaxval[nthreads];
        
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
        unsigned int idx = min(tid, gridDim.x * gridDim.y - 1);

        sminval[tid] = minval[idx];
        smaxval[tid] = maxval[idx];
        __syncthreads();

		find_min_max_in_smem<nthreads, best_type>(sminval, smaxval, tid);

        if (tid == 0) 
        {
            minval[0] = (T)sminval[0];
            maxval[0] = (T)smaxval[0];
            blocks_finished = 0;
        }
    }


    template <typename T>
    void min_max_mask_multipass_caller(const DevMem2D src, const PtrStep mask, double* minval, double* maxval, PtrStep buf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        T* minval_buf = (T*)buf.ptr(0);
        T* maxval_buf = (T*)buf.ptr(1);

        min_max_kernel<256, T, Mask8U><<<grid, threads>>>(src, Mask8U(mask), minval_buf, maxval_buf);
        min_max_pass2_kernel<256, T><<<1, 256>>>(minval_buf, maxval_buf, grid.x * grid.y);
        cudaSafeCall(hipDeviceSynchronize());

        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;
    }

    template void min_max_mask_multipass_caller<unsigned char>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_multipass_caller<char>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_multipass_caller<unsigned short>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_multipass_caller<short>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_multipass_caller<int>(const DevMem2D, const PtrStep, double*, double*, PtrStep);
    template void min_max_mask_multipass_caller<float>(const DevMem2D, const PtrStep, double*, double*, PtrStep);


    template <typename T>
    void min_max_multipass_caller(const DevMem2D src, double* minval, double* maxval, PtrStep buf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        T* minval_buf = (T*)buf.ptr(0);
        T* maxval_buf = (T*)buf.ptr(1);

        min_max_kernel<256, T, MaskTrue><<<grid, threads>>>(src, MaskTrue(), minval_buf, maxval_buf);
        min_max_pass2_kernel<256, T><<<1, 256>>>(minval_buf, maxval_buf, grid.x * grid.y);
        cudaSafeCall(hipDeviceSynchronize());

        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;
    }

    template void min_max_multipass_caller<unsigned char>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_multipass_caller<char>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_multipass_caller<unsigned short>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_multipass_caller<short>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_multipass_caller<int>(const DevMem2D, double*, double*, PtrStep);
    template void min_max_multipass_caller<float>(const DevMem2D, double*, double*, PtrStep);

    } // namespace minmax

///////////////////////////////////////////////////////////////////////////////
// minMaxLoc

    namespace minmaxloc {

    __constant__ int ctwidth;
    __constant__ int ctheight;

    // Global counter of blocks finished its work
    __device__ unsigned int blocks_finished = 0;


    // Estimates good thread configuration
    //  - threads variable satisfies to threads.x * threads.y == 256
    void estimate_thread_cfg(int cols, int rows, dim3& threads, dim3& grid)
    {
        threads = dim3(32, 8);
        grid = dim3(divUp(cols, threads.x * 8), divUp(rows, threads.y * 32));
    }


    // Returns required buffer sizes
    void get_buf_size_required(int cols, int rows, int elem_size, int& b1cols, 
                               int& b1rows, int& b2cols, int& b2rows)
    {
        dim3 threads, grid;
        estimate_thread_cfg(cols, rows, threads, grid);
        b1cols = grid.x * grid.y * elem_size; // For values
        b1rows = 2;
        b2cols = grid.x * grid.y * sizeof(int); // For locations
        b2rows = 2;
    }


    // Estimates device constants which are used in the kernels using specified thread configuration
    void set_kernel_consts(int cols, int rows, const dim3& threads, const dim3& grid)
    {        
        int twidth = divUp(divUp(cols, grid.x), threads.x);
        int theight = divUp(divUp(rows, grid.y), threads.y);
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctwidth), &twidth, sizeof(ctwidth))); 
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctheight), &theight, sizeof(ctheight))); 
    }  


    template <typename T>
    __device__ void merge(unsigned int tid, unsigned int offset, volatile T* minval, volatile T* maxval, 
                          volatile unsigned int* minloc, volatile unsigned int* maxloc)
    {
        T val = minval[tid + offset];
        if (val < minval[tid])
        {
            minval[tid] = val;
            minloc[tid] = minloc[tid + offset];
        }
        val = maxval[tid + offset];
        if (val > maxval[tid])
        {
            maxval[tid] = val;
            maxloc[tid] = maxloc[tid + offset];
        }
    }


    template <int size, typename T>
    __device__ void find_min_max_loc_in_smem(volatile T* minval, volatile T* maxval, volatile unsigned int* minloc, 
                                             volatile unsigned int* maxloc, const unsigned int tid)
    {
        if (size >= 512) { if (tid < 256) { merge(tid, 256, minval, maxval, minloc, maxloc); } __syncthreads(); }
        if (size >= 256) { if (tid < 128) { merge(tid, 128, minval, maxval, minloc, maxloc); }  __syncthreads(); }
        if (size >= 128) { if (tid < 64) { merge(tid, 64, minval, maxval, minloc, maxloc); } __syncthreads(); }

        if (tid < 32)
        {
            if (size >= 64) merge(tid, 32, minval, maxval, minloc, maxloc);
            if (size >= 32) merge(tid, 16, minval, maxval, minloc, maxloc);
            if (size >= 16) merge(tid, 8, minval, maxval, minloc, maxloc);
            if (size >= 8) merge(tid, 4, minval, maxval, minloc, maxloc);
            if (size >= 4) merge(tid, 2, minval, maxval, minloc, maxloc);
            if (size >= 2) merge(tid, 1, minval, maxval, minloc, maxloc);
        }
    }


    template <int nthreads, typename T, typename Mask>
    __global__ void min_max_loc_kernel(const DevMem2D src, Mask mask, T* minval, T* maxval, 
                                       unsigned int* minloc, unsigned int* maxloc)
    {
        typedef typename MinMaxTypeTraits<T>::best_type best_type;
        __shared__ best_type sminval[nthreads];
        __shared__ best_type smaxval[nthreads];
        __shared__ unsigned int sminloc[nthreads];
        __shared__ unsigned int smaxloc[nthreads];

        unsigned int x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
        unsigned int y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        T mymin = numeric_limits_gpu<T>::max();
        T mymax = numeric_limits_gpu<T>::is_signed ? -numeric_limits_gpu<T>::max() : numeric_limits_gpu<T>::min(); 
        unsigned int myminloc = 0;
        unsigned int mymaxloc = 0;
        unsigned int y_end = min(y0 + (ctheight - 1) * blockDim.y + 1, src.rows);
        unsigned int x_end = min(x0 + (ctwidth - 1) * blockDim.x + 1, src.cols);

        for (unsigned int y = y0; y < y_end; y += blockDim.y)
        {
            const T* ptr = (const T*)src.ptr(y);
            for (unsigned int x = x0; x < x_end; x += blockDim.x)
            {
                if (mask(y, x))
                {
                    T val = ptr[x];
                    if (val <= mymin) { mymin = val; myminloc = y * src.cols + x; }
                    if (val >= mymax) { mymax = val; mymaxloc = y * src.cols + x; }
                }
            }
        }

        sminval[tid] = mymin; 
        smaxval[tid] = mymax;
        sminloc[tid] = myminloc;
        smaxloc[tid] = mymaxloc;
        __syncthreads();

        find_min_max_loc_in_smem<nthreads, best_type>(sminval, smaxval, sminloc, smaxloc, tid);

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 110
		__shared__ bool is_last;

		if (tid == 0)
		{
			minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
            maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
            minloc[blockIdx.y * gridDim.x + blockIdx.x] = sminloc[0];
            maxloc[blockIdx.y * gridDim.x + blockIdx.x] = smaxloc[0];
			__threadfence();

			unsigned int ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
			is_last = ticket == gridDim.x * gridDim.y - 1;
		}

		__syncthreads();

		if (is_last)
		{
            unsigned int idx = min(tid, gridDim.x * gridDim.y - 1);

            sminval[tid] = minval[idx];
            smaxval[tid] = maxval[idx];
            sminloc[tid] = minloc[idx];
            smaxloc[tid] = maxloc[idx];
            __syncthreads();

			find_min_max_loc_in_smem<nthreads, best_type>(sminval, smaxval, sminloc, smaxloc, tid);

            if (tid == 0) 
            {
                minval[0] = (T)sminval[0];
                maxval[0] = (T)smaxval[0];
                minloc[0] = sminloc[0];
                maxloc[0] = smaxloc[0];
                blocks_finished = 0;
            }
		}
#else
        if (tid == 0) 
        {
            minval[blockIdx.y * gridDim.x + blockIdx.x] = (T)sminval[0];
            maxval[blockIdx.y * gridDim.x + blockIdx.x] = (T)smaxval[0];
            minloc[blockIdx.y * gridDim.x + blockIdx.x] = sminloc[0];
            maxloc[blockIdx.y * gridDim.x + blockIdx.x] = smaxloc[0];
        }
#endif
    }


    template <typename T>
    void min_max_loc_mask_caller(const DevMem2D src, const PtrStep mask, double* minval, double* maxval, 
                                 int minloc[2], int maxloc[2], PtrStep valbuf, PtrStep locbuf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        T* minval_buf = (T*)valbuf.ptr(0);
        T* maxval_buf = (T*)valbuf.ptr(1);
        unsigned int* minloc_buf = (unsigned int*)locbuf.ptr(0);
        unsigned int* maxloc_buf = (unsigned int*)locbuf.ptr(1);

        min_max_loc_kernel<256, T, Mask8U><<<grid, threads>>>(src, Mask8U(mask), minval_buf, maxval_buf, minloc_buf, maxloc_buf);
        cudaSafeCall(hipDeviceSynchronize());

        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;

        unsigned int minloc_, maxloc_;
        cudaSafeCall(hipMemcpy(&minloc_, minloc_buf, sizeof(int), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxloc_, maxloc_buf, sizeof(int), hipMemcpyDeviceToHost));
        minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
        maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
    }

    template void min_max_loc_mask_caller<unsigned char>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_caller<char>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_caller<unsigned short>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_caller<short>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_caller<int>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_caller<float>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_caller<double>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);


    template <typename T>
    void min_max_loc_caller(const DevMem2D src, double* minval, double* maxval, 
                            int minloc[2], int maxloc[2], PtrStep valbuf, PtrStep locbuf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        T* minval_buf = (T*)valbuf.ptr(0);
        T* maxval_buf = (T*)valbuf.ptr(1);
        unsigned int* minloc_buf = (unsigned int*)locbuf.ptr(0);
        unsigned int* maxloc_buf = (unsigned int*)locbuf.ptr(1);

        min_max_loc_kernel<256, T, MaskTrue><<<grid, threads>>>(src, MaskTrue(), minval_buf, maxval_buf, minloc_buf, maxloc_buf);
        cudaSafeCall(hipDeviceSynchronize());

        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;

        unsigned int minloc_, maxloc_;
        cudaSafeCall(hipMemcpy(&minloc_, minloc_buf, sizeof(int), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxloc_, maxloc_buf, sizeof(int), hipMemcpyDeviceToHost));
        minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
        maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
    }

    template void min_max_loc_caller<unsigned char>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_caller<char>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_caller<unsigned short>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_caller<short>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_caller<int>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_caller<float>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_caller<double>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);


    // This kernel will be used only when compute capability is 1.0
    template <int nthreads, typename T>
    __global__ void min_max_loc_pass2_kernel(T* minval, T* maxval, unsigned int* minloc, unsigned int* maxloc, int size)
    {
        typedef typename MinMaxTypeTraits<T>::best_type best_type;
        __shared__ best_type sminval[nthreads];
        __shared__ best_type smaxval[nthreads];
        __shared__ unsigned int sminloc[nthreads];
        __shared__ unsigned int smaxloc[nthreads];

        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
        unsigned int idx = min(tid, gridDim.x * gridDim.y - 1);

        sminval[tid] = minval[idx];
        smaxval[tid] = maxval[idx];
        sminloc[tid] = minloc[idx];
        smaxloc[tid] = maxloc[idx];
        __syncthreads();

		find_min_max_loc_in_smem<nthreads, best_type>(sminval, smaxval, sminloc, smaxloc, tid);

        if (tid == 0) 
        {
            minval[0] = (T)sminval[0];
            maxval[0] = (T)smaxval[0];
            minloc[0] = sminloc[0];
            maxloc[0] = smaxloc[0];
            blocks_finished = 0;
        }
    }


    template <typename T>
    void min_max_loc_mask_multipass_caller(const DevMem2D src, const PtrStep mask, double* minval, double* maxval, 
                                           int minloc[2], int maxloc[2], PtrStep valbuf, PtrStep locbuf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        T* minval_buf = (T*)valbuf.ptr(0);
        T* maxval_buf = (T*)valbuf.ptr(1);
        unsigned int* minloc_buf = (unsigned int*)locbuf.ptr(0);
        unsigned int* maxloc_buf = (unsigned int*)locbuf.ptr(1);

        min_max_loc_kernel<256, T, Mask8U><<<grid, threads>>>(src, Mask8U(mask), minval_buf, maxval_buf, minloc_buf, maxloc_buf);
        min_max_loc_pass2_kernel<256, T><<<1, 256>>>(minval_buf, maxval_buf, minloc_buf, maxloc_buf, grid.x * grid.y);
        cudaSafeCall(hipDeviceSynchronize());

        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;

        unsigned int minloc_, maxloc_;
        cudaSafeCall(hipMemcpy(&minloc_, minloc_buf, sizeof(int), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxloc_, maxloc_buf, sizeof(int), hipMemcpyDeviceToHost));
        minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
        maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
    }

    template void min_max_loc_mask_multipass_caller<unsigned char>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_multipass_caller<char>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_multipass_caller<unsigned short>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_multipass_caller<short>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_multipass_caller<int>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_mask_multipass_caller<float>(const DevMem2D, const PtrStep, double*, double*, int[2], int[2], PtrStep, PtrStep);


    template <typename T>
    void min_max_loc_multipass_caller(const DevMem2D src, double* minval, double* maxval, 
                                      int minloc[2], int maxloc[2], PtrStep valbuf, PtrStep locbuf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        T* minval_buf = (T*)valbuf.ptr(0);
        T* maxval_buf = (T*)valbuf.ptr(1);
        unsigned int* minloc_buf = (unsigned int*)locbuf.ptr(0);
        unsigned int* maxloc_buf = (unsigned int*)locbuf.ptr(1);

        min_max_loc_kernel<256, T, MaskTrue><<<grid, threads>>>(src, MaskTrue(), minval_buf, maxval_buf, minloc_buf, maxloc_buf);
        min_max_loc_pass2_kernel<256, T><<<1, 256>>>(minval_buf, maxval_buf, minloc_buf, maxloc_buf, grid.x * grid.y);
        cudaSafeCall(hipDeviceSynchronize());

        T minval_, maxval_;
        cudaSafeCall(hipMemcpy(&minval_, minval_buf, sizeof(T), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxval_, maxval_buf, sizeof(T), hipMemcpyDeviceToHost));
        *minval = minval_;
        *maxval = maxval_;

        unsigned int minloc_, maxloc_;
        cudaSafeCall(hipMemcpy(&minloc_, minloc_buf, sizeof(int), hipMemcpyDeviceToHost));
        cudaSafeCall(hipMemcpy(&maxloc_, maxloc_buf, sizeof(int), hipMemcpyDeviceToHost));
        minloc[1] = minloc_ / src.cols; minloc[0] = minloc_ - minloc[1] * src.cols;
        maxloc[1] = maxloc_ / src.cols; maxloc[0] = maxloc_ - maxloc[1] * src.cols;
    }

    template void min_max_loc_multipass_caller<unsigned char>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_multipass_caller<char>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_multipass_caller<unsigned short>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_multipass_caller<short>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_multipass_caller<int>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);
    template void min_max_loc_multipass_caller<float>(const DevMem2D, double*, double*, int[2], int[2], PtrStep, PtrStep);

    } // namespace minmaxloc

//////////////////////////////////////////////////////////////////////////////////////////////////////////
// countNonZero

    namespace countnonzero 
    {

    __constant__ int ctwidth;
    __constant__ int ctheight;

    __device__ unsigned int blocks_finished = 0;

    void estimate_thread_cfg(int cols, int rows, dim3& threads, dim3& grid)
    {
        threads = dim3(32, 8);
        grid = dim3(divUp(cols, threads.x * 8), divUp(rows, threads.y * 32));
    }


    void get_buf_size_required(int cols, int rows, int& bufcols, int& bufrows)
    {
        dim3 threads, grid;
        estimate_thread_cfg(cols, rows, threads, grid);
        bufcols = grid.x * grid.y * sizeof(int);
        bufrows = 1;
    }


    void set_kernel_consts(int cols, int rows, const dim3& threads, const dim3& grid)
    {        
        int twidth = divUp(divUp(cols, grid.x), threads.x);
        int theight = divUp(divUp(rows, grid.y), threads.y);
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctwidth), &twidth, sizeof(twidth))); 
        cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(ctheight), &theight, sizeof(theight))); 
    }


    template <int size, typename T>
    __device__ void sum_is_smem(volatile T* data, const unsigned int tid)
    {
        T sum = data[tid];

        if (size >= 512) { if (tid < 256) { data[tid] = sum = sum + data[tid + 256]; } __syncthreads(); }
        if (size >= 256) { if (tid < 128) { data[tid] = sum = sum + data[tid + 128]; } __syncthreads(); }
        if (size >= 128) { if (tid < 64) { data[tid] = sum = sum + data[tid + 64]; } __syncthreads(); }

        if (tid < 32)
        {
            if (size >= 64) data[tid] = sum = sum + data[tid + 32];
            if (size >= 32) data[tid] = sum = sum + data[tid + 16];
            if (size >= 16) data[tid] = sum = sum + data[tid + 8];
            if (size >= 8) data[tid] = sum = sum + data[tid + 4];
            if (size >= 4) data[tid] = sum = sum + data[tid + 2];
            if (size >= 2) data[tid] = sum = sum + data[tid + 1];
        }
    }


    template <int nthreads, typename T>
    __global__ void count_non_zero_kernel(const DevMem2D src, volatile unsigned int* count)
    {
        __shared__ unsigned int scount[nthreads];

        unsigned int x0 = blockIdx.x * blockDim.x * ctwidth + threadIdx.x;
        unsigned int y0 = blockIdx.y * blockDim.y * ctheight + threadIdx.y;
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

		unsigned int cnt = 0;
        for (unsigned int y = 0; y < ctheight && y0 + y * blockDim.y < src.rows; ++y)
        {
            const T* ptr = (const T*)src.ptr(y0 + y * blockDim.y);
            for (unsigned int x = 0; x < ctwidth && x0 + x * blockDim.x < src.cols; ++x)
				cnt += ptr[x0 + x * blockDim.x] != 0;
		}

		scount[tid] = cnt;
		__syncthreads();

        sum_is_smem<nthreads, unsigned int>(scount, tid);

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 110
		__shared__ bool is_last;

		if (tid == 0)
		{
			count[blockIdx.y * gridDim.x + blockIdx.x] = scount[0];
			__threadfence();

			unsigned int ticket = atomicInc(&blocks_finished, gridDim.x * gridDim.y);
			is_last = ticket == gridDim.x * gridDim.y - 1;
		}

		__syncthreads();

		if (is_last)
		{
            scount[tid] = tid < gridDim.x * gridDim.y ? count[tid] : 0;
            __syncthreads();

			sum_is_smem<nthreads, unsigned int>(scount, tid);

			if (tid == 0) 
            {
                count[0] = scount[0];
                blocks_finished = 0;
            }
		}
#else
        if (tid == 0) count[blockIdx.y * gridDim.x + blockIdx.x] = scount[0];
#endif
    }

   
    template <typename T>
    int count_non_zero_caller(const DevMem2D src, PtrStep buf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        unsigned int* count_buf = (unsigned int*)buf.ptr(0);

        count_non_zero_kernel<256, T><<<grid, threads>>>(src, count_buf);
        cudaSafeCall(hipDeviceSynchronize());

        unsigned int count;
        cudaSafeCall(hipMemcpy(&count, count_buf, sizeof(int), hipMemcpyDeviceToHost));
        
        return count;
    }  

    template int count_non_zero_caller<unsigned char>(const DevMem2D, PtrStep);
    template int count_non_zero_caller<char>(const DevMem2D, PtrStep);
    template int count_non_zero_caller<unsigned short>(const DevMem2D, PtrStep);
    template int count_non_zero_caller<short>(const DevMem2D, PtrStep);
    template int count_non_zero_caller<int>(const DevMem2D, PtrStep);
    template int count_non_zero_caller<float>(const DevMem2D, PtrStep);
    template int count_non_zero_caller<double>(const DevMem2D, PtrStep);


    template <int nthreads, typename T>
    __global__ void count_non_zero_pass2_kernel(unsigned int* count, int size)
    {
        __shared__ unsigned int scount[nthreads];
        unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

        scount[tid] = tid < size ? count[tid] : 0;
		sum_is_smem<nthreads, unsigned int>(scount, tid);

		if (tid == 0) 
        {
            count[0] = scount[0];
            blocks_finished = 0;
        }
    }


    template <typename T>
    int count_non_zero_multipass_caller(const DevMem2D src, PtrStep buf)
    {
        dim3 threads, grid;
        estimate_thread_cfg(src.cols, src.rows, threads, grid);
        set_kernel_consts(src.cols, src.rows, threads, grid);

        unsigned int* count_buf = (unsigned int*)buf.ptr(0);

        count_non_zero_kernel<256, T><<<grid, threads>>>(src, count_buf);
        count_non_zero_pass2_kernel<256, T><<<1, 256>>>(count_buf, grid.x * grid.y);
        cudaSafeCall(hipDeviceSynchronize());

        unsigned int count;
        cudaSafeCall(hipMemcpy(&count, count_buf, sizeof(int), hipMemcpyDeviceToHost));
        
        return count;
    }  

    template int count_non_zero_multipass_caller<unsigned char>(const DevMem2D, PtrStep);
    template int count_non_zero_multipass_caller<char>(const DevMem2D, PtrStep);
    template int count_non_zero_multipass_caller<unsigned short>(const DevMem2D, PtrStep);
    template int count_non_zero_multipass_caller<short>(const DevMem2D, PtrStep);
    template int count_non_zero_multipass_caller<int>(const DevMem2D, PtrStep);
    template int count_non_zero_multipass_caller<float>(const DevMem2D, PtrStep);

    } // namespace countnonzero

//////////////////////////////////////////////////////////////////////////////////////////////////////////
// transpose

    template <typename T>
    __global__ void transpose(const DevMem2D_<T> src, PtrStep_<T> dst)
    {
    	__shared__ T s_mem[16 * 17];

    	int x = blockIdx.x * blockDim.x + threadIdx.x;
    	int y = blockIdx.y * blockDim.y + threadIdx.y;
	    int smem_idx = threadIdx.y * blockDim.x + threadIdx.x + threadIdx.y;

	    if (y < src.rows && x < src.cols)
	    {
            s_mem[smem_idx] = src.ptr(y)[x];
	    }
	    __syncthreads();

	    smem_idx = threadIdx.x * blockDim.x + threadIdx.y + threadIdx.x;

	    x = blockIdx.y * blockDim.x + threadIdx.x;
	    y = blockIdx.x * blockDim.y + threadIdx.y;

	    if (y < src.cols && x < src.rows)
	    {
		    dst.ptr(y)[x] = s_mem[smem_idx];
	    }
    }

    template <typename T>
    void transpose_gpu(const DevMem2D& src, const DevMem2D& dst)
    {
	    dim3 threads(16, 16, 1);
	    dim3 grid(divUp(src.cols, 16), divUp(src.rows, 16), 1);

	    transpose<T><<<grid, threads>>>((DevMem2D_<T>)src, (DevMem2D_<T>)dst);
        cudaSafeCall( hipDeviceSynchronize() );
    }

    template void transpose_gpu<uchar4 >(const DevMem2D& src, const DevMem2D& dst);
    template void transpose_gpu<char4  >(const DevMem2D& src, const DevMem2D& dst);
    template void transpose_gpu<ushort2>(const DevMem2D& src, const DevMem2D& dst);
    template void transpose_gpu<short2 >(const DevMem2D& src, const DevMem2D& dst);
    template void transpose_gpu<int    >(const DevMem2D& src, const DevMem2D& dst);
    template void transpose_gpu<float  >(const DevMem2D& src, const DevMem2D& dst);
}}}
