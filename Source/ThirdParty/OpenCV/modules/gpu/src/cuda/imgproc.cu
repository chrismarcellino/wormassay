#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include "cuda_shared.hpp"
#include "border_interpolate.hpp"

using namespace cv::gpu;

/////////////////////////////////// Remap ///////////////////////////////////////////////
namespace cv { namespace gpu { namespace imgproc
{
    texture<unsigned char, 2, hipReadModeNormalizedFloat> tex_remap;

    __global__ void remap_1c(const float* mapx, const float* mapy, size_t map_step, uchar* out, size_t out_step, int width, int height)
    {    
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        if (x < width && y < height)
        {
            int idx = y * (map_step >> 2) + x; /* map_step >> 2  <=> map_step / sizeof(float)*/

            float xcoo = mapx[idx];
            float ycoo = mapy[idx];

            out[y * out_step + x] = (unsigned char)(255.f * tex2D(tex_remap, xcoo, ycoo));            
        }
    }

    __global__ void remap_3c(const uchar* src, size_t src_step, const float* mapx, const float* mapy, size_t map_step, 
                             uchar* dst, size_t dst_step, int width, int height)
    {    
        const int x = blockDim.x * blockIdx.x + threadIdx.x;
        const int y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x < width && y < height)
        {
            const int idx = y * (map_step >> 2) + x; /* map_step >> 2  <=> map_step / sizeof(float)*/

            const float xcoo = mapx[idx];
            const float ycoo = mapy[idx];
            
            uchar3 out = make_uchar3(0, 0, 0);

            if (xcoo >= 0 && xcoo < width - 1 && ycoo >= 0 && ycoo < height - 1)
            {
                const int x1 = __float2int_rd(xcoo);
                const int y1 = __float2int_rd(ycoo);
                const int x2 = x1 + 1;
                const int y2 = y1 + 1;
                
                uchar src_reg = *(src + y1 * src_step + 3 * x1);
                out.x += src_reg * (x2 - xcoo) * (y2 - ycoo);
                src_reg = *(src + y1 * src_step + 3 * x1 + 1);
                out.y += src_reg * (x2 - xcoo) * (y2 - ycoo);
                src_reg = *(src + y1 * src_step + 3 * x1 + 2);
                out.z += src_reg * (x2 - xcoo) * (y2 - ycoo);

                src_reg = *(src + y1 * src_step + 3 * x2);                
                out.x += src_reg * (xcoo - x1) * (y2 - ycoo);
                src_reg = *(src + y1 * src_step + 3 * x2 + 1); 
                out.y += src_reg * (xcoo - x1) * (y2 - ycoo);
                src_reg = *(src + y1 * src_step + 3 * x2 + 2); 
                out.z += src_reg * (xcoo - x1) * (y2 - ycoo);

                src_reg = *(src + y2 * src_step + 3 * x1);                
                out.x += src_reg * (x2 - xcoo) * (ycoo - y1);
                src_reg = *(src + y2 * src_step + 3 * x1 + 1); 
                out.y += src_reg * (x2 - xcoo) * (ycoo - y1);
                src_reg = *(src + y2 * src_step + 3 * x1 + 2); 
                out.z += src_reg * (x2 - xcoo) * (ycoo - y1);

                src_reg = *(src + y2 * src_step + 3 * x2);                
                out.x += src_reg * (xcoo - x1) * (ycoo - y1);
                src_reg = *(src + y2 * src_step + 3 * x2 + 1);  
                out.y += src_reg * (xcoo - x1) * (ycoo - y1);
                src_reg = *(src + y2 * src_step + 3 * x2 + 2);  
                out.z += src_reg * (xcoo - x1) * (ycoo - y1);
            }

            /**(uchar3*)(dst + y * dst_step + 3 * x) = out;*/
            *(dst + y * dst_step + 3 * x) = out.x;
            *(dst + y * dst_step + 3 * x + 1) = out.y;
            *(dst + y * dst_step + 3 * x + 2) = out.z;
        }
    }

    void remap_gpu_1c(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, DevMem2D dst)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(dst.cols, threads.x);
        grid.y = divUp(dst.rows, threads.y);

        tex_remap.filterMode = hipFilterModeLinear;	    
        tex_remap.addressMode[0] = tex_remap.addressMode[1] = hipAddressModeWrap;
        hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
        cudaSafeCall( hipBindTexture2D(0, tex_remap, src.data, desc, src.cols, src.rows, src.step) );

        remap_1c<<<grid, threads>>>(xmap.data, ymap.data, xmap.step, dst.data, dst.step, dst.cols, dst.rows);

        cudaSafeCall( hipDeviceSynchronize() );  
        cudaSafeCall( hipUnbindTexture(tex_remap) );
    }
    
    void remap_gpu_3c(const DevMem2D& src, const DevMem2Df& xmap, const DevMem2Df& ymap, DevMem2D dst)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(dst.cols, threads.x);
        grid.y = divUp(dst.rows, threads.y);

        remap_3c<<<grid, threads>>>(src.data, src.step, xmap.data, ymap.data, xmap.step, dst.data, dst.step, dst.cols, dst.rows);

        cudaSafeCall( hipDeviceSynchronize() ); 
    }

/////////////////////////////////// MeanShiftfiltering ///////////////////////////////////////////////

    texture<uchar4, 2> tex_meanshift;

    __device__ short2 do_mean_shift(int x0, int y0, unsigned char* out, 
                                    int out_step, int cols, int rows, 
                                    int sp, int sr, int maxIter, float eps)
    {
        int isr2 = sr*sr;
        uchar4 c = tex2D(tex_meanshift, x0, y0 );

        // iterate meanshift procedure
        for( int iter = 0; iter < maxIter; iter++ )
        {
            int count = 0;
            int s0 = 0, s1 = 0, s2 = 0, sx = 0, sy = 0;
            float icount;

            //mean shift: process pixels in window (p-sigmaSp)x(p+sigmaSp)
            int minx = x0-sp;
            int miny = y0-sp;
            int maxx = x0+sp;
            int maxy = y0+sp;

            for( int y = miny; y <= maxy; y++)
            {
                int rowCount = 0;
                for( int x = minx; x <= maxx; x++ )
                {                    
                    uchar4 t = tex2D( tex_meanshift, x, y );

                    int norm2 = (t.x - c.x) * (t.x - c.x) + (t.y - c.y) * (t.y - c.y) + (t.z - c.z) * (t.z - c.z);
                    if( norm2 <= isr2 )
                    {
                        s0 += t.x; s1 += t.y; s2 += t.z;
                        sx += x; rowCount++;
                    }
                }
                count += rowCount;
                sy += y*rowCount;
            }

            if( count == 0 )
                break;

            icount = 1.f/count;
            int x1 = __float2int_rz(sx*icount);
            int y1 = __float2int_rz(sy*icount);
            s0 = __float2int_rz(s0*icount);
            s1 = __float2int_rz(s1*icount);
            s2 = __float2int_rz(s2*icount);

            int norm2 = (s0 - c.x) * (s0 - c.x) + (s1 - c.y) * (s1 - c.y) + (s2 - c.z) * (s2 - c.z);

            bool stopFlag = (x0 == x1 && y0 == y1) || (abs(x1-x0) + abs(y1-y0) + norm2 <= eps);

            x0 = x1; y0 = y1;
            c.x = s0; c.y = s1; c.z = s2;

            if( stopFlag )
                break;
        }

        int base = (blockIdx.y * blockDim.y + threadIdx.y) * out_step + (blockIdx.x * blockDim.x + threadIdx.x) * 4 * sizeof(uchar);
        *(uchar4*)(out + base) = c;

        return make_short2((short)x0, (short)y0);
    }

    extern "C" __global__ void meanshift_kernel( unsigned char* out, int out_step, int cols, int rows, 
                                                 int sp, int sr, int maxIter, float eps )
    {
        int x0 = blockIdx.x * blockDim.x + threadIdx.x;
        int y0 = blockIdx.y * blockDim.y + threadIdx.y;

        if( x0 < cols && y0 < rows )
            do_mean_shift(x0, y0, out, out_step, cols, rows, sp, sr, maxIter, eps);
    }

    extern "C" __global__ void meanshiftproc_kernel( unsigned char* outr, int outrstep, 
                                                 unsigned char* outsp, int outspstep, 
                                                 int cols, int rows, 
                                                 int sp, int sr, int maxIter, float eps )
    {
        int x0 = blockIdx.x * blockDim.x + threadIdx.x;
        int y0 = blockIdx.y * blockDim.y + threadIdx.y;

        if( x0 < cols && y0 < rows )
        {            
            int basesp = (blockIdx.y * blockDim.y + threadIdx.y) * outspstep + (blockIdx.x * blockDim.x + threadIdx.x) * 2 * sizeof(short);
            *(short2*)(outsp + basesp) = do_mean_shift(x0, y0, outr, outrstep, cols, rows, sp, sr, maxIter, eps);
        }
    }

    extern "C" void meanShiftFiltering_gpu(const DevMem2D& src, DevMem2D dst, int sp, int sr, int maxIter, float eps)
    {                        
        dim3 grid(1, 1, 1);
        dim3 threads(32, 16, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

        meanshift_kernel<<< grid, threads >>>( dst.data, dst.step, dst.cols, dst.rows, sp, sr, maxIter, eps );
        cudaSafeCall( hipDeviceSynchronize() );
        cudaSafeCall( hipUnbindTexture( tex_meanshift ) );        
    }
    extern "C" void meanShiftProc_gpu(const DevMem2D& src, DevMem2D dstr, DevMem2D dstsp, int sp, int sr, int maxIter, float eps) 
    {
        dim3 grid(1, 1, 1);
        dim3 threads(32, 16, 1);
        grid.x = divUp(src.cols, threads.x);
        grid.y = divUp(src.rows, threads.y);

        hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
        cudaSafeCall( hipBindTexture2D( 0, tex_meanshift, src.data, desc, src.cols, src.rows, src.step ) );

        meanshiftproc_kernel<<< grid, threads >>>( dstr.data, dstr.step, dstsp.data, dstsp.step, dstr.cols, dstr.rows, sp, sr, maxIter, eps );
        cudaSafeCall( hipDeviceSynchronize() );
        cudaSafeCall( hipUnbindTexture( tex_meanshift ) );        
    }

/////////////////////////////////// drawColorDisp ///////////////////////////////////////////////

    template <typename T>
    __device__ unsigned int cvtPixel(T d, int ndisp, float S = 1, float V = 1)
    {        
        unsigned int H = ((ndisp-d) * 240)/ndisp;

        unsigned int hi = (H/60) % 6;
        float f = H/60.f - H/60;
        float p = V * (1 - S);
        float q = V * (1 - f * S);
        float t = V * (1 - (1 - f) * S);

        float3 res;
        
        if (hi == 0) //R = V,	G = t,	B = p
        {
            res.x = p;
            res.y = t;
            res.z = V;
        }

        if (hi == 1) // R = q,	G = V,	B = p
        {
            res.x = p;
            res.y = V;
            res.z = q;
        }        
        
        if (hi == 2) // R = p,	G = V,	B = t
        {
            res.x = t;
            res.y = V;
            res.z = p;
        }
            
        if (hi == 3) // R = p,	G = q,	B = V
        {
            res.x = V;
            res.y = q;
            res.z = p;
        }

        if (hi == 4) // R = t,	G = p,	B = V
        {
            res.x = V;
            res.y = p;
            res.z = t;
        }

        if (hi == 5) // R = V,	G = p,	B = q
        {
            res.x = q;
            res.y = p;
            res.z = V;
        }
        const unsigned int b = (unsigned int)(max(0.f, min (res.x, 1.f)) * 255.f);
        const unsigned int g = (unsigned int)(max(0.f, min (res.y, 1.f)) * 255.f);
        const unsigned int r = (unsigned int)(max(0.f, min (res.z, 1.f)) * 255.f);
        const unsigned int a = 255U;

        return (a << 24) + (r << 16) + (g << 8) + b;    
    } 

    __global__ void drawColorDisp(uchar* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) 
        {
            uchar4 d4 = *(uchar4*)(disp + y * disp_step + x);

            uint4 res;
            res.x = cvtPixel(d4.x, ndisp);
            res.y = cvtPixel(d4.y, ndisp);
            res.z = cvtPixel(d4.z, ndisp);
            res.w = cvtPixel(d4.w, ndisp);
                    
            uint4* line = (uint4*)(out_image + y * out_step);
            line[x >> 2] = res;
        }
    }

    __global__ void drawColorDisp(short* disp, size_t disp_step, uchar* out_image, size_t out_step, int width, int height, int ndisp)
    {
        const int x = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if(x < width && y < height) 
        {
            short2 d2 = *(short2*)(disp + y * disp_step + x);

            uint2 res;
            res.x = cvtPixel(d2.x, ndisp);            
            res.y = cvtPixel(d2.y, ndisp);

            uint2* line = (uint2*)(out_image + y * out_step);
            line[x >> 1] = res;
        }
    }


    void drawColorDisp_gpu(const DevMem2D& src, const DevMem2D& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(16, 16, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 2);
        grid.y = divUp(src.rows, threads.y);
         
        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step, dst.data, dst.step, src.cols, src.rows, ndisp);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() ); 
    }

    void drawColorDisp_gpu(const DevMem2D_<short>& src, const DevMem2D& dst, int ndisp, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(src.cols, threads.x << 1);
        grid.y = divUp(src.rows, threads.y);
         
        drawColorDisp<<<grid, threads, 0, stream>>>(src.data, src.step / sizeof(short), dst.data, dst.step, src.cols, src.rows, ndisp);
        
        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

/////////////////////////////////// reprojectImageTo3D ///////////////////////////////////////////////

    __constant__ float cq[16];

    template <typename T>
    __global__ void reprojectImageTo3D(const T* disp, size_t disp_step, float* xyzw, size_t xyzw_step, int rows, int cols)
    {        
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (y < rows && x < cols)
        {

            float qx = cq[1] * y + cq[3], qy = cq[5] * y + cq[7];
            float qz = cq[9] * y + cq[11], qw = cq[13] * y + cq[15];

            qx += x * cq[0]; 
            qy += x * cq[4];
            qz += x * cq[8];
            qw += x * cq[12];

            T d = *(disp + disp_step * y + x);

            float iW = 1.f / (qw + cq[14] * d);
            float4 v;
            v.x = (qx + cq[2] * d) * iW;
            v.y = (qy + cq[6] * d) * iW;
            v.z = (qz + cq[10] * d) * iW;
            v.w = 1.f;

            *(float4*)(xyzw + xyzw_step * y + (x * 4)) = v;
        }
    }

    template <typename T>
    inline void reprojectImageTo3D_caller(const DevMem2D_<T>& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        dim3 threads(32, 8, 1);
        dim3 grid(1, 1, 1);
        grid.x = divUp(disp.cols, threads.x);
        grid.y = divUp(disp.rows, threads.y);

        cudaSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(cq), q, 16 * sizeof(float)) );

        reprojectImageTo3D<<<grid, threads, 0, stream>>>(disp.data, disp.step / sizeof(T), xyzw.data, xyzw.step / sizeof(float), disp.rows, disp.cols);

        if (stream == 0)
            cudaSafeCall( hipDeviceSynchronize() );
    }

    void reprojectImageTo3D_gpu(const DevMem2D& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        reprojectImageTo3D_caller(disp, xyzw, q, stream);
    }

    void reprojectImageTo3D_gpu(const DevMem2D_<short>& disp, const DevMem2Df& xyzw, const float* q, const hipStream_t& stream)
    {
        reprojectImageTo3D_caller(disp, xyzw, q, stream);
    }

//////////////////////////////////////// Extract Cov Data ////////////////////////////////////////////////

    __global__ void extractCovData_kernel(const int cols, const int rows, const PtrStepf Dx, 
                                          const PtrStepf Dy, PtrStepf dst)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {            
            float dx = Dx.ptr(y)[x];
            float dy = Dy.ptr(y)[x];

            dst.ptr(y)[x] = dx * dx;
            dst.ptr(y + rows)[x] = dx * dy;
            dst.ptr(y + (rows << 1))[x] = dy * dy;
        }
    }

    void extractCovData_caller(const DevMem2Df Dx, const DevMem2Df Dy, PtrStepf dst)
    {
        dim3 threads(32, 8);
        dim3 grid(divUp(Dx.cols, threads.x), divUp(Dx.rows, threads.y));

        extractCovData_kernel<<<grid, threads>>>(Dx.cols, Dx.rows, Dx, Dy, dst);
        cudaSafeCall(hipDeviceSynchronize());
    }

/////////////////////////////////////////// Corner Harris /////////////////////////////////////////////////

    texture<float, 2> harrisDxTex;
    texture<float, 2> harrisDyTex;

    template <typename B>
    __global__ void cornerHarris_kernel(const int cols, const int rows, const int block_size, const float k,
                                        PtrStep dst, B border_row, B border_col)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                int y = border_col.idx(i);
                for (int j = jbegin; j < jend; ++j)
                {
                    int x = border_row.idx(j);
                    float dx = tex2D(harrisDxTex, x, y);
                    float dy = tex2D(harrisDyTex, x, y);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            ((float*)dst.ptr(y))[x] = a * c - b * b - k * (a + c) * (a + c);
        }
    }

    void cornerHarris_caller(const int block_size, const float k, const DevMem2D Dx, const DevMem2D Dy, DevMem2D dst, 
                             int border_type)
    {
        const int rows = Dx.rows;
        const int cols = Dx.cols;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
        hipBindTexture2D(0, harrisDxTex, Dx.data, desc, Dx.cols, Dx.rows, Dx.step);
        hipBindTexture2D(0, harrisDyTex, Dy.data, desc, Dy.cols, Dy.rows, Dy.step);
        harrisDxTex.filterMode = hipFilterModePoint;
        harrisDyTex.filterMode = hipFilterModePoint;

        switch (border_type) 
        {
        case BORDER_REFLECT101:
            cornerHarris_kernel<<<grid, threads>>>(
                    cols, rows, block_size, k, dst, BrdReflect101(cols), BrdReflect101(rows));
            break;
        }

        cudaSafeCall(hipDeviceSynchronize());
        cudaSafeCall(hipUnbindTexture(harrisDxTex));
        cudaSafeCall(hipUnbindTexture(harrisDyTex));
    }

/////////////////////////////////////////// Corner Min Eigen Val /////////////////////////////////////////////////

    texture<float, 2> minEigenValDxTex;
    texture<float, 2> minEigenValDyTex;

    template <typename B>
    __global__ void cornerMinEigenVal_kernel(const int cols, const int rows, const int block_size, 
                                             PtrStep dst, B border_row, B border_col)
    {
        const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float a = 0.f;
            float b = 0.f;
            float c = 0.f;

            const int ibegin = y - (block_size / 2);
            const int jbegin = x - (block_size / 2);
            const int iend = ibegin + block_size;
            const int jend = jbegin + block_size;

            for (int i = ibegin; i < iend; ++i)
            {
                int y = border_col.idx(i);
                for (int j = jbegin; j < jend; ++j)
                {
                    int x = border_row.idx(j);
                    float dx = tex2D(minEigenValDxTex, x, y);
                    float dy = tex2D(minEigenValDyTex, x, y);
                    a += dx * dx;
                    b += dx * dy;
                    c += dy * dy;
                }
            }

            a *= 0.5f;
            c *= 0.5f;
            ((float*)dst.ptr(y))[x] = (a + c) - sqrtf((a - c) * (a - c) + b * b);
        }
    }

    void cornerMinEigenVal_caller(const int block_size, const DevMem2D Dx, const DevMem2D Dy, DevMem2D dst,
                                  int border_type)
    {
        const int rows = Dx.rows;
        const int cols = Dx.cols;

        dim3 threads(32, 8);
        dim3 grid(divUp(cols, threads.x), divUp(rows, threads.y));

        hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
        hipBindTexture2D(0, minEigenValDxTex, Dx.data, desc, Dx.cols, Dx.rows, Dx.step);
        hipBindTexture2D(0, minEigenValDyTex, Dy.data, desc, Dy.cols, Dy.rows, Dy.step);
        minEigenValDxTex.filterMode = hipFilterModePoint;
        minEigenValDyTex.filterMode = hipFilterModePoint;

        switch (border_type)
        {
        case BORDER_REFLECT101:
            cornerMinEigenVal_kernel<<<grid, threads>>>(
                    cols, rows, block_size, dst, 
                    BrdReflect101(cols), BrdReflect101(rows));
            break;
        }

        cudaSafeCall(hipDeviceSynchronize());
        cudaSafeCall(hipUnbindTexture(minEigenValDxTex));
        cudaSafeCall(hipUnbindTexture(minEigenValDyTex));
    }
}}}

